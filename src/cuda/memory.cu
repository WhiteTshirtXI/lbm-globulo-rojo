
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void alloc_memory_GPU(int X, int Y, int Z, float **cells_d, float **flags_d, float **vel_d, float **rho_d, float **fuerza_d) {

	gpuErrchk( hipMalloc(cells_d, 2*X*Y*Z*19*sizeof(float)) );
	gpuErrchk( hipMalloc(flags_d, X*Y*Z*sizeof(float)) );
	gpuErrchk( hipMalloc(vel_d, X*Y*Z*3*sizeof(float)) );
	gpuErrchk( hipMalloc(rho_d, X*Y*Z*sizeof(float)) );
	gpuErrchk( hipMalloc(fuerza_d, X*Y*Z*3*sizeof(float)) );

}

void free_memory_GPU(float *cells_d, float *flags_d, float *vel_d, float *rho_d, float *fuerza_d) {

	gpuErrchk( hipFree(cells_d) );
	gpuErrchk( hipFree(flags_d) );
	gpuErrchk( hipFree(vel_d) );
	gpuErrchk( hipFree(rho_d) );
	gpuErrchk( hipFree(fuerza_d) );

}

void send_data_to_GPU(int X, int Y, int Z, float *cells, float *cells_d, float *flags, float *flags_d, float *vel, float *vel_d, float *rho, float *rho_d, float *fuerza, float *fuerza_d) {

	gpuErrchk( hipMemcpy(cells_d, cells, 2*X*Y*Z*19*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(flags_d, flags, X*Y*Z*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(vel_d, vel, X*Y*Z*3*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(rho_d, rho, X*Y*Z*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(fuerza_d, fuerza, X*Y*Z*3*sizeof(float), hipMemcpyHostToDevice) );

}

void retrieve_data_from_GPU(int X, int Y, int Z, float *cells, float *cells_d, float *flags, float *flags_d, float *vel, float *vel_d, float *rho, float *rho_d, float *fuerza, float *fuerza_d) {

	gpuErrchk( hipMemcpy(cells, cells_d, 2*X*Y*Z*19*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(flags, flags_d, X*Y*Z*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(vel, vel_d, X*Y*Z*3*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(rho, rho_d, X*Y*Z*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(fuerza, fuerza_d, X*Y*Z*3*sizeof(float), hipMemcpyDeviceToHost) );

}
