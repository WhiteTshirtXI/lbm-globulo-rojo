
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void alloc_memory_GPU(float **data_d, size_t size)
{
	gpuErrchk( hipMalloc(data_d, size) );

}

void free_memory_GPU(float *cells_d, float *flags_d, float *vel_d, float *rho_d, float *fuerza_d, float *vertex_d, float *velocidad_d, float *velocidad2_d) {

	gpuErrchk( hipFree(cells_d) );
	gpuErrchk( hipFree(flags_d) );
	gpuErrchk( hipFree(vel_d) );
	gpuErrchk( hipFree(rho_d) );
	gpuErrchk( hipFree(fuerza_d) );

	gpuErrchk( hipFree(vertex_d) );
	gpuErrchk( hipFree(velocidad_d) );
	gpuErrchk( hipFree(velocidad2_d) );
}

void send_data_to_GPU(int X, int Y, int Z, float *cells, float *cells_d, float *flags, float *flags_d, float *vel, float *vel_d, float *rho, float *rho_d, float *fuerza, float *fuerza_d, int nNodos, float *vertex, float *vertex_d, float *velocidad, float *velocidad_d, float *velocidad2, float *velocidad2_d) {

	gpuErrchk( hipMemcpy(cells_d, cells, 2*X*Y*Z*19*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(flags_d, flags, X*Y*Z*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(vel_d, vel, X*Y*Z*3*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(rho_d, rho, X*Y*Z*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(fuerza_d, fuerza, X*Y*Z*3*sizeof(float), hipMemcpyHostToDevice) );

	gpuErrchk( hipMemcpy(vertex_d, vertex, nNodos*3*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(velocidad_d, velocidad, nNodos*3*sizeof(float), hipMemcpyHostToDevice) );
	gpuErrchk( hipMemcpy(velocidad2_d, velocidad2, nNodos*3*sizeof(float), hipMemcpyHostToDevice) );

}

void retrieve_data_from_GPU(int X, int Y, int Z, float *cells, float *cells_d, float *flags, float *flags_d, float *vel, float *vel_d, float *rho, float *rho_d, float *fuerza, float *fuerza_d, int nNodos, float *vertex, float *vertex_d, float *velocidad, float *velocidad_d, float *velocidad2, float *velocidad2_d) {

	gpuErrchk( hipMemcpy(cells, cells_d, 2*X*Y*Z*19*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(flags, flags_d, X*Y*Z*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(vel, vel_d, X*Y*Z*3*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(rho, rho_d, X*Y*Z*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(fuerza, fuerza_d, X*Y*Z*3*sizeof(float), hipMemcpyDeviceToHost) );

	gpuErrchk( hipMemcpy(vertex, vertex_d, nNodos*3*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(velocidad, velocidad_d, nNodos*3*sizeof(float), hipMemcpyDeviceToHost) );
	gpuErrchk( hipMemcpy(velocidad2, velocidad2_d, nNodos*3*sizeof(float), hipMemcpyDeviceToHost) );

}
