
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

void alloc_memory_GPU(void **data_d, size_t size)
{
	gpuErrchk( hipMalloc(data_d, size) );

}

void free_memory_GPU(void *data_d) {

	gpuErrchk( hipFree(data_d) );
}

void send_data_to_GPU(void *data, void *data_d, size_t size) {

	gpuErrchk( hipMemcpy(data_d, data, size, hipMemcpyHostToDevice) );

}

void retrieve_data_from_GPU(void *data, void *data_d, size_t size) {

	gpuErrchk( hipMemcpy(data, data_d, size, hipMemcpyDeviceToHost) );

}
