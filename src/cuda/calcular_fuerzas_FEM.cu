#include "hip/hip_runtime.h"
#include "helper.h"

__device__ void MdotV_d(float M[3][3], float V[3], float *p)
{
	p[0] = M[0][0]*V[0] + M[0][1]*V[1] + M[0][2]*V[2];
	p[1] = M[1][0]*V[0] + M[1][1]*V[1] + M[1][2]*V[2];
	p[2] = M[2][0]*V[0] + M[2][1]*V[1] + M[2][2]*V[2];
}

__device__ float norm_d_bis(float a[3])
{
	return sqrt( (a[0]*a[0]) + (a[1]*a[1]) + (a[2]*a[2]));
}

__device__ void fuerzas_d(float referencia[3][3], float deformado[3][3], float fuerzas[3][3], float ks)
{
	// Coordenadas de los tres vertices iniciales
	float xi=referencia[0][0];
	float yi=referencia[0][1];
	//float zi=referencia[0][2];

	float xj=referencia[1][0];
	float yj=referencia[1][1];
	//float zj=referencia[1][2];

	float xk=referencia[2][0];
	float yk=referencia[2][1];
	//float zk=referencia[2][2];

	// Coordenadas de los tres vertices del elemento deformado
	float Xi=deformado[0][0];
	float Yi=deformado[0][1];
	//float Zi=deformado[0][2];

	float Xj=deformado[1][0];
	float Yj=deformado[1][1];
	//float Zj=deformado[1][2];

	float Xk=deformado[2][0];
	float Yk=deformado[2][1];
	//float Zk=deformado[2][2];

	// Desplazamientos de cada nodo
	float ui=Xi-xi;
	float vi=Yi-yi;

	float uj=Xj-xj;
	float vj=Yj-yj;

	float uk=Xk-xk;
	float vk=Yk-yk;

	// Coeficientes de funciones de forma
	float ai = yj-yk;
	float bi = xk-xj;
	float ci = xj*yk - xk*yj;
	float Li = ai*xi + bi*yi + ci;

	float aj = yk - yi;
	float bj = xi - xk;
	float cj = xk*yi - xi*yk;
	float Lj = aj*xj + bj*yj + cj;

	float ak = yi - yj;
	float bk = xj - xi;
	float ck = xi*yj - xj*yi;
	float Lk = ak*xk + bk*yk + ck;

	// Derivadas parciales para calcular el vector [G] revision Sep/14/2010
	float dudx = ui*ai/Li + uj*aj/Lj + uk*ak/Lk;
	float dudy = ui*bi/Li + uj*bj/Lj + uk*bk/Lk;
	float dvdx = vi*ai/Li + vj*aj/Lj + vk*ak/Lk;
	float dvdy = vi*bi/Li + vj*bj/Lj + vk*bk/Lk;

	// Componentes del vector [G] revision Sep/14/2010
	float g11 = (1.0+dudx)*(1.0+dudx) + (dvdx)*(dvdx);
	float g12 = (1.0+dudx)*(dudy) + (1.0+dvdy)*(dvdx);
	//float g21 = g12;
	float g22 = (1.0+dvdy)*(1.0+dvdy) + (dudy)*(dudy);
	//float G[2][2] = {{g11,g12},{g21,g22}};

	// Calculo de lambda1 y lambda2 revision Sep/14/2010
	float l1 = sqrt((g11 + g22 + sqrt((g11-g22)*(g11-g22) + 4.*g12*g12))/2.);
	float l2 = sqrt((g11 + g22 - sqrt((g11-g22)*(g11-g22) + 4.*g12*g12))/2.);

	// Derivadas de la funcion Strain Energy respecto a lambda 1 y lambda 2
	// Modelo de energia Skalak 1973 revision Sep/14/2010
	float I1 = (l1*l1) + (l2*l2) - 2.0;
	float I2 = (l1*l1)*(l2*l2) - 1.0;
	float dI1dl1 = 2.0*l1;
	float dI1dl2 = 2.0*l2;
	float dI2dl1 = 2.0*l1*(l2*l2);
	float dI2dl2 = 2.0*l2*(l1*l1);

	/*
				TODO Aqui se cambia la funcion de densidad de energia
	 */
	//    dwdl1 = (B/4)*(I1*dI1dl1 + dI1dl1 - dI2dl1) + (C/4)*(I2)*(dI2dl1)
	//    dwdl2 = (B/4)*(I1*dI1dl2 + dI1dl2 - dI2dl2) + (C/4)*(I2)*(dI2dl2)
	float dwdl1 = (ks/12.)*(2.*I1*dI1dl1 + 2.0*dI1dl1 - 2.0*dI2dl1) + (ks/6.0)*I2*dI2dl1;
	float dwdl2 = (ks/12.)*(2.*I1*dI1dl2 + 2.0*dI1dl2 - 2.0*dI2dl2) + (ks/6.0)*I2*dI2dl2;

	// Calculo de diferenciales sobre l1 y l2 respecto a desplazamientos de nodos
	// 1. Derivadas de [G] respecto a desplazamiento de nodos revision Sep/14/2010
	float dg11dui = 2.0*(1.0+ dudx)*(ai/Li);
	float dg11duj = 2.0*(1.0+ dudx)*(aj/Lj);
	float dg11duk = 2.0*(1.0+ dudx)*(ak/Lk);
	float dg11dvi = 2.0*dvdx*(ai/Li);
	float dg11dvj = 2.0*dvdx*(aj/Lj);
	float dg11dvk = 2.0*dvdx*(ak/Lk);

	float dg12dui = (1.0+dudx)*(bi/Li) + (ai/Li)*(dudy);
	float dg12duj = (1.0+dudx)*(bj/Lj) + (aj/Lj)*(dudy);
	float dg12duk = (1.0+dudx)*(bk/Lk) + (ak/Lk)*(dudy);
	float dg12dvi = (1.0+dvdy)*(ai/Li) + (bi/Li)*(dvdx);
	float dg12dvj = (1.0+dvdy)*(aj/Lj) + (bj/Lj)*(dvdx);
	float dg12dvk = (1.0+dvdy)*(ak/Lk) + (bk/Lk)*(dvdx);

	float dg22dui = 2.0*dudy*(bi/Li);
	float dg22duj = 2.0*dudy*(bj/Lj);
	float dg22duk = 2.0*dudy*(bk/Lk);

	float dg22dvi = 2.0*(1.0+dvdy)*(bi/Li);
	float dg22dvj = 2.0*(1.0+dvdy)*(bj/Lj);
	float dg22dvk = 2.0*(1.0+dvdy)*(bk/Lk);

	/*
				TODO Revisar implementacion de dlambada/du
	 */
	// 2. Calculo de las derivadas de lambda 1 y lambda 2 respecto desplazamientos
	// nodales revision Sep/14/2010
	float t0 = sqrt((g11-g22)*(g11-g22) + 4.*g12*g12);

	// Derivadas de lambda1  y lambda 2

	float dt0dui;
	if(t0>10e-3){
		dt0dui = ((((g11-g22)*(dg11dui-dg22dui))+(4.*g12*dg12dui))/(t0));}
	else{
		dt0dui = 0.0;}
	float dl1dui = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11dui + dg22dui + dt0dui);
	float dl2dui = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11dui + dg22dui - dt0dui);

	float dt0duj;
	if(t0>10e-3){
		dt0duj = ((((g11-g22)*(dg11duj-dg22duj))+(4.*g12*dg12duj))/(t0));}
	else{
		dt0duj = 0.0;}
	float dl1duj = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11duj + dg22duj + dt0duj);
	float dl2duj = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11duj + dg22duj - dt0duj);

	float dt0duk;
	if(t0>10e-3){
		dt0duk = ((((g11-g22)*(dg11duk-dg22duk))+(4.*g12*dg12duk))/(t0));}
	else{
		dt0duk = 0.0;}
	float dl1duk = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11duk + dg22duk + dt0duk);
	float dl2duk = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11duk + dg22duk - dt0duk);

	float dt0dvi;
	if(t0>10e-3){
		dt0dvi = ((((g11-g22)*(dg11dvi-dg22dvi))+(4.*g12*dg12dvi))/(t0));}
	else{
		dt0dvi = 0.0;}
	float dl1dvi = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11dvi + dg22dvi + dt0dvi);
	float dl2dvi = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11dvi + dg22dvi - dt0dvi);

	float dt0dvj;
	if(t0>10e-3){
		dt0dvj = ((((g11-g22)*(dg11dvj-dg22dvj))+(4.*g12*dg12dvj))/(t0));}
	else{
		dt0dvj = 0.0;}
	float dl1dvj = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11dvj + dg22dvj + dt0dvj);
	float dl2dvj = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11dvj + dg22dvj - dt0dvj);

	float dt0dvk;
	if(t0>10e-3){
		dt0dvk = ((((g11-g22)*(dg11dvk-dg22dvk))+(4.*g12*dg12dvk))/(t0));}
	else{
		dt0dvk = 0.0;}
	float dl1dvk = ((sqrt(0.5)*0.5)/(sqrt(g11+g22+t0)))*(dg11dvk + dg22dvk + dt0dvk);
	float dl2dvk = ((sqrt(0.5)*0.5)/(sqrt(g11+g22-t0)))*(dg11dvk + dg22dvk - dt0dvk);

	// 3. Calculo de las derivadas de w respecto a los desplazamientos nodales
	// revision Sep/14/2010
	float dwdui = dwdl1*dl1dui + dwdl2*dl2dui;
	float dwdvi = dwdl1*dl1dvi + dwdl2*dl2dvi;
	float dwduj = dwdl1*dl1duj + dwdl2*dl2duj;
	float dwdvj = dwdl1*dl1dvj + dwdl2*dl2dvj;
	float dwduk = dwdl1*dl1duk + dwdl2*dl2duk;
	float dwdvk = dwdl1*dl1dvk + dwdl2*dl2dvk;

	// 4. Volumen del elemento revision Sep/14/2010
	float a0 = ((xj-xi)*(yk-yi) + (xk-xi)*(yj-yi))/2.0;
	float espesor = 0.5;//0.0025;

	// 5. Calculo de las componentes de fuerza revision Sep/14/2010
	float fxi = dwdui*a0*espesor;
	float fyi = dwdvi*a0*espesor;
	float fzi = 0.0;
	float fxj = dwduj*a0*espesor;
	float fyj = dwdvj*a0*espesor;
	float fzj = 0.0;
	float fxk = dwduk*a0*espesor;
	float fyk = dwdvk*a0*espesor;
	float fzk = 0.0;

	fuerzas[0][0] = fxi;
	fuerzas[0][1] = fyi;
	fuerzas[0][2] = fzi;
	fuerzas[1][0] = fxj;
	fuerzas[1][1] = fyj;
	fuerzas[1][2] = fzj;
	fuerzas[2][0] = fxk;
	fuerzas[2][1] = fyk;
	fuerzas[2][2] = fzk;
}


__device__ void cross_d(float a[3], float b[3], float &x, float &y, float &z)
{
	x = a[1]*b[2] - a[2]*b[1];
	y = a[2]*b[0] - a[0]*b[2];
	z = a[0]*b[1] - a[1]*b[0];
}

__device__ void rotacion_d(float referencia[3][3], float deformado[3][3], float nfuerzas[3][3], float ks)
{
	// Coordenadas iniciales de los tres nodos respecto al sistema
	// global de coordenadas i1, i2, i3
	float xi = referencia[0][0];
	float yi = referencia[0][1];
	float zi = referencia[0][2];

	float xj = referencia[1][0];
	float yj = referencia[1][1];
	float zj = referencia[1][2];

	float xk = referencia[2][0];
	float yk = referencia[2][1];
	float zk = referencia[2][2];

	// Coordenadas del elemento deformado de los tres nodos
	// respecto al sistema global de coordenadas i1, i2, i3
	float Xi = deformado[0][0];
	float Yi = deformado[0][1];
	float Zi = deformado[0][2];

	float Xj = deformado[1][0];
	float Yj = deformado[1][1];
	float Zj = deformado[1][2];

	float Xk = deformado[2][0];
	float Yk = deformado[2][1];
	float Zk = deformado[2][2];

	// Unit vectors Local undeformed local coordinate axis
	float m1 = sqrt((xj-xi)*(xj-xi) + (yj-yi)*(yj-yi) + (zj-zi)*(zj-zi));
	float ve1[3] = {(xj-xi)/m1, (yj-yi)/m1, (zj-zi)/m1};
	float m2 = sqrt((xk-xi)*(xk-xi) + (yk-yi)*(yk-yi) + (zk-zi)*(zk-zi));
	float ve4[3] = {(xk-xi)/m2, (yk-yi)/m2, (zk-zi)/m2};
	float ve3[3];
	cross_d(ve1, ve4, ve3[0], ve3[1], ve3[2]);
	float m3 = norm_d_bis(ve3);

	ve3[0] = ve3[0]/m3;
	ve3[1] = ve3[1]/m3;
	ve3[2] = ve3[2]/m3;

	float ve2[3];
	cross_d(ve3,ve1,ve2[0], ve2[1], ve2[2]);

	// Formar la base e
	float e[3][3];
	e[0][0] = ve1[0];
	e[0][1] = ve1[1];
	e[0][2] = ve1[2];

	e[1][0] = ve2[0];
	e[1][1] = ve2[1];
	e[1][2] = ve2[2];

	e[2][0] = ve3[0];
	e[2][1] = ve3[1];
	e[2][2] = ve3[2];

	// Matriz de rotacion para la configuracion NO deformada
	// [r] hacia las coordenadas globales
	float d1 = (xj-xi)/m1;
	float d2 = (yj-yi)/m1;
	float d3 = (zj-zi)/m1;
	float e1 = (xk-xi)/m2;
	float e2 = (yk-yi)/m2;
	float e3 = (zk-zi)/m2;
	float f1 = (d2*e3-d3*e2)/m3;
	float f2 = (d3*e1-d1*e3)/m3;
	float f3 = (d1*e2-d2*e1)/m3;
	float g1 = f2*d3 - f3*d2;
	float g2 = f3*d1 - f1*d3;
	float g3 = f1*d2 - f2*d1;
	float r[3][3] = {{d1, d2, d3},{g1, g2, g3},{f1, f2, f3}};

	// Unit vectors Local deformed coordinate axis
	float M1 = sqrt((Xj-Xi)*(Xj-Xi) + (Yj-Yi)*(Yj-Yi) + (Zj-Zi)*(Zj-Zi));
	float E1[3] = {(Xj-Xi)/M1, (Yj-Yi)/M1, (Zj-Zi)/M1};
	float M2 = sqrt((Xk-Xi)*(Xk-Xi) + (Yk-Yi)*(Yk-Yi) + (Zk-Zi)*(Zk-Zi));
	float E4[4] = {(Xk-Xi)/M2, (Yk-Yi)/M2, (Zk-Zi)/M2};
	float E3[3];
	cross_d(E1,E4,E3[0], E3[1], E3[2]);
	float M3 = norm_d_bis(E3);
	E3[0] = E3[0]/M3;
	E3[1] = E3[1]/M3;
	E3[2] = E3[2]/M3;
	float E2[3];
	cross_d(E3,E1,E2[0],E2[1],E2[2]);

	// Formar la base E
	float E[3][3];
	E[0][0] = E1[0];
	E[0][1] = E1[1];
	E[0][2] = E1[2];

	E[1][0] = E2[0];
	E[1][1] = E2[1];
	E[1][2] = E2[2];

	E[2][0] = E3[0];
	E[2][1] = E3[1];
	E[2][2] = E3[2];

	// Matriz de rotacion para la configuracion deformada
	// [R] hacia la base global i
	d1 = (Xj-Xi)/M1;
	d2 = (Yj-Yi)/M1;
	d3 = (Zj-Zi)/M1;
	e1 = (Xk-Xi)/M2;
	e2 = (Yk-Yi)/M2;
	e3 = (Zk-Zi)/M2;
	f1 = (d2*e3-d3*e2)/M3;
	f2 = (d3*e1-d1*e3)/M3;
	f3 = (d1*e2-d2*e1)/M3;
	g1 = f2*d3 - f3*d2;
	g2 = f3*d1 - f1*d3;
	g3 = f1*d2 - f2*d1;
	float R[3][3] = {{d1, d2, d3},{g1, g2, g3},{f1, f2, f3}};

	// Vectores posicion de los nodos en coordenadas locales estado deformado
	// tiene origen en el nodo i deformado
	float Xil = 0.0;
	float Yil = 0.0;
	float Zil = 0.0;
	float Xjl = Xj - Xi;
	float Yjl = Yj - Yi;
	float Zjl = Zj - Zi;
	float Xkl = Xk - Xi;
	float Ykl = Yk - Yi;
	float Zkl = Zk - Zi;


	// Transformar cada coordenada local del elemento deformado
	// a la base comun i,j,k
	float Pi[3];
	float v1[3] = {Xil, Yil, Zil};
	MdotV_d(R, v1, Pi);

	float Pj[3];
	float v2[3]={Xjl, Yjl, Zjl};
	MdotV_d(R,v2, Pj);

	float Pk[3];
	float v3[3]={Xkl, Ykl, Zkl};
	MdotV_d(R,v3, Pk);

	// Vectores posicion de los nodos en coordenadas locales estado NO deformado
	// tiene origen en el nodo i NO deformado
	float xil = 0.0;
	float yil = 0.0;
	float zil = 0.0;
	float xjl = xj - xi;
	float yjl = yj - yi ;
	float zjl = zj - zi;
	float xkl = xk - xi;
	float ykl = yk - yi;
	float zkl = zk - zi;

	// Transformar cada coordenada local del elemento sin deformar a la base i,j,k
	float V1[3] = {xil, yil, zil};
	float pi[3];
	MdotV_d(r,V1, pi);

	float V2[3] = {xjl, yjl, zjl};
	float pj[3];
	MdotV_d(r,V2, pj);

	float V3[3] = {xkl, ykl, zkl};
	float pk[3];
	MdotV_d(r,V3, pk);

	// Vectores de desplazamientos en el plano
	float di[3];
	di[0] = Pi[0] - pi[0];
	di[1] = Pi[1] - pi[1];
	di[2] = Pi[2] - pi[2];

	float dj[3];
	dj[0] = Pj[0] - pj[0];
	dj[1] = Pj[1] - pj[1];
	dj[2] = Pj[2] - pj[2];

	float dk[3];
	dk[0] = Pk[0] - pk[0];
	dk[1] = Pk[1] - pk[1];
	dk[2] = Pk[2] - pk[2];

	// Llama rutina para calcular las fuerzas en el elemento deformado
	float ref[3][3], def[3][3];
	ref[0][0] = pi[0];
	ref[0][1] = pi[1];
	ref[0][2] = pi[2];

	ref[1][0] = pj[0];
	ref[1][1] = pj[1];
	ref[1][2] = pj[2];

	ref[2][0] = pk[0];
	ref[2][1] = pk[1];
	ref[2][2] = pk[2];

	def[0][0] = Pi[0];
	def[0][1] = Pi[1];
	def[0][2] = Pi[2];

	def[1][0] = Pj[0];
	def[1][1] = Pj[1];
	def[1][2] = Pj[2];

	def[2][0] = Pk[0];
	def[2][1] = Pk[1];
	def[2][2] = Pk[2];
	fuerzas_d(ref, def, nfuerzas, ks);

	// Transformar las fuerzas calculadas a la base original
	float tR[3][3];
	for(int i = 0; i<3; i++ )
		for(int j = 0; j<3; j++ )
		{
			tR[i][j] = R[j][i];
		}
	float nfn1[3], fn1[3] = {nfuerzas[0][0], nfuerzas[0][1], nfuerzas[0][2]};
	float nfn2[3], fn2[3] = {nfuerzas[1][0], nfuerzas[1][1], nfuerzas[1][2]};
	float nfn3[3], fn3[3] = {nfuerzas[2][0], nfuerzas[2][1], nfuerzas[2][2]};
	MdotV_d(tR, fn1, nfn1);
	MdotV_d(tR, fn2, nfn2);
	MdotV_d(tR, fn3, nfn3);

	nfuerzas[0][0] = nfn1[0];
	nfuerzas[0][1] = nfn1[1];
	nfuerzas[0][2] = nfn1[2];

	nfuerzas[1][0] = nfn2[0];
	nfuerzas[1][1] = nfn2[1];
	nfuerzas[1][2] = nfn2[2];

	nfuerzas[2][0] = nfn3[0];
	nfuerzas[2][1] = nfn3[1];
	nfuerzas[2][2] = nfn3[2];
}


__global__ void calcular_fuerzas_FEM(int nNodos, int nCeldas, int *faces_d, float *vertex_d, float *vertex_ref_d, float ks, float *fuerza_mesh_d) {

	int e = blockIdx.x*blockDim.x + threadIdx.x;

	if (e < nCeldas) {

		int a,b,c;
		float va[3], vb[3], vc[3], vA[3], vB[3], vC[3];
		// Crear estructuras para pasar al algoritmo de fuerzas
		float ref[3][3], def[3][3], fuerzas[3][3];

		// Obtener indices de los nodos de cada elemento
		a = FACES_D(e, 0);
		b = FACES_D(e, 1);
		c = FACES_D(e, 2);

		// Obtener las posiciones de cada vertice no deformado
		va[0] = VERTEX_REF_D(a, 0);
		va[1] = VERTEX_REF_D(a, 1);
		va[2] = VERTEX_REF_D(a, 2);

		vb[0] = VERTEX_REF_D(b, 0);
		vb[1] = VERTEX_REF_D(b, 1);
		vb[2] = VERTEX_REF_D(b, 2);

		vc[0] = VERTEX_REF_D(c, 0);
		vc[1] = VERTEX_REF_D(c, 1);
		vc[2] = VERTEX_REF_D(c, 2);

		// Obtener las posiciones de cada vertice deformado
		vA[0] = VERTEX_D(a, 0);
		vA[1] = VERTEX_D(a, 1);
		vA[2] = VERTEX_D(a, 2);

		vB[0] = VERTEX_D(b, 0);
		vB[1] = VERTEX_D(b, 1);
		vB[2] = VERTEX_D(b, 2);

		vC[0] = VERTEX_D(c, 0);
		vC[1] = VERTEX_D(c, 1);
		vC[2] = VERTEX_D(c, 2);


		ref[0][0] = va[0];
		ref[0][1] = va[1];
		ref[0][2] = va[2];

		ref[1][0] = vb[0];
		ref[1][1] = vb[1];
		ref[1][2] = vb[2];

		ref[2][0] = vc[0];
		ref[2][1] = vc[1];
		ref[2][2] = vc[2];

		def[0][0] = vA[0];
		def[0][1] = vA[1];
		def[0][2] = vA[2];

		def[1][0] = vB[0];
		def[1][1] = vB[1];
		def[1][2] = vB[2];

		def[2][0] = vC[0];
		def[2][1] = vC[1];
		def[2][2] = vC[2];

		rotacion_d(ref, def, fuerzas, ks);

		//Agregar cada fuerza a cada nodo
		FUERZA_MESH_D(a, 0) = FUERZA_MESH_D(a, 0)-fuerzas[0][0];
		FUERZA_MESH_D(a, 1) = FUERZA_MESH_D(a, 1)-fuerzas[0][1];
		FUERZA_MESH_D(a, 2) = FUERZA_MESH_D(a, 2)-fuerzas[0][2];

		FUERZA_MESH_D(b, 0) = FUERZA_MESH_D(b, 0)-fuerzas[1][0];
		FUERZA_MESH_D(b, 1) = FUERZA_MESH_D(b, 1)-fuerzas[1][1];
		FUERZA_MESH_D(b, 2) = FUERZA_MESH_D(b, 2)-fuerzas[1][2];

		FUERZA_MESH_D(c, 0) = FUERZA_MESH_D(c, 0)-fuerzas[2][0];
		FUERZA_MESH_D(c, 1) = FUERZA_MESH_D(c, 1)-fuerzas[2][1];
		FUERZA_MESH_D(c, 2) = FUERZA_MESH_D(c, 2)-fuerzas[2][2];

	}
}


void calcular_fuerzas_FEM_wrapper(int nNodos, int nCeldas, int *faces_d, float *vertex_d, float *vertex_ref_d, float ks, float *fuerza_mesh_d) {


	//X*Y*Z = 9261;
	//Maximum number of threads per block:           1024

	dim3 grid_size;
	grid_size.x = nCeldas/1024 + 1;


	dim3 block_size;
	// 1000 threads per blocks
	block_size.x = 1024;

	//Launch kernel
	calcular_fuerzas_FEM<<<grid_size, block_size>>>(nNodos, nCeldas, faces_d, vertex_d, vertex_ref_d, ks, fuerza_mesh_d);


}
